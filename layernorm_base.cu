#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <cmath>
#include <stdio.h>
#include <time.h>

#define CUDA_CHECK(call) \
if ((call) != hipSuccess) { \
fprintf(stderr, "CUDA error: %s at %s:%d\n", hipGetErrorString(call), __FILE__, __LINE__); \
exit(1); \
}

void rand(float* elements, int size) {
    for(int i = 0; i < size; i++){
        elements[i] = (float)rand() / RAND_MAX;
    }
}

bool check(float *out, float *res, int n) {
    for (int i = 0; i < n; i++) {
        if (std::fabs(out[i] - res[i]) > 1e-5) {
            return false;
        }
    }
    return true;
}

/**
 * int B int T int C 输入/输出的 shape (8,1024,768)
 * const float* inp 输入向量
 * float* mean float* rstd 均值和标准差的倒数  (8,1024)
 * const float* weight const float* bias 权重和偏置 (768)
 * float* out 输出向量
 */
void layernorm_cpu(const float* inp, float* mean, float* rstd, const float* weight, const float* bias, float* out
    , int B, int T, int C);

void layernorm_gpu(const float* inp, float* mean, float* rstd, const float* weight, const float* bias, float* out
    , int B, int T, int C, int block_size);

int main(int argc, char *argv[]) {
    int B = 64, T = 1024, C = 768, block_size = 128;
    float* inp = (float*) malloc(B * T * C * sizeof(float));
    rand(inp, B * T * C);

    float* mean = (float*) malloc(B * T * sizeof(float));
    float* rstd = (float*) malloc(B * T * sizeof(float));

    float* d_mean = (float*) malloc(B * T * sizeof(float));
    float* d_rstd = (float*) malloc(B * T * sizeof(float));

    float* weight = (float*) malloc(C * sizeof(float));
    rand(weight, C);
    float* bias = (float*) malloc(C * sizeof(float));
    rand(bias, C);

    float* out = (float*) malloc(B * T * C * sizeof(float));
    float* d_out = (float*) malloc(B * T * C * sizeof(float));

    clock_t start = clock();
    layernorm_cpu(inp, mean, rstd, weight, bias, out, B, T, C);
    clock_t end = clock();
    double cpu_time = (double)(end - start) / CLOCKS_PER_SEC;
    printf("CPU耗时: %.6f ms\n", cpu_time * 1000);

    layernorm_gpu(inp, d_mean, d_rstd, weight, bias, d_out, B, T, C, block_size);

    if (check(mean, d_mean, B * T) && check(rstd, d_rstd, B * T) && check(out, d_out, B * T * C)) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
    }

    free(inp);
    free(out);
    free(mean);
    free(rstd);
    free(weight);
    free(bias);
}

__global__ void layernorm_kernel(const float* inp, float* mean, float* rstd, const float* weight, const float* bias, float* out
    , int B, int T, int C) {
    float eps = 1e-5f;
    //计算当前word的位置
    const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < B * T) {
        //计算均值
        float m = 0.0f;
        for (int c = 0; c < C; c++) {
            m += inp[tid * C + c];
        }
        m = m / C;
        //计算方差
        float v = 0.0f;
        for (int c = 0; c < C; c++) {
            float x_i = inp[tid * C + c] - m;
            v += x_i * x_i;
        }
        v = v / C;
        //计算标准差的倒数
        float s = 1.0f / sqrtf(v + eps);
        //计算结果输出
        for (int c = 0; c < C; c++) {
            float n = s * (inp[tid * C + c] - m);
            out[tid * C + c] = weight[c] * n + bias[c];
        }
        //记录均值和标准差倒数
        mean[tid] = m;
        rstd[tid] = s;
    }
}

void layernorm_gpu(const float *inp, float *mean, float *rstd, const float *weight, const float *bias, float *out, int B, int T, int C, int block_size) {
    //计算grid
    double N = B * T;
    int grid_size = ceil((N + block_size - 1)/ block_size);

    //分配显存
    float* d_inp;
    hipMalloc(&d_inp, B * T * C * sizeof(float));

    float* d_mean;
    hipMalloc(&d_mean, B * T * sizeof(float));

    float* d_rstd;
    hipMalloc(&d_rstd, B * T * sizeof(float));

    float* d_weight;
    hipMalloc(&d_weight, C * sizeof(float));

    float* d_bias;
    hipMalloc(&d_bias, C * sizeof(float));

    float* d_out;
    hipMalloc(&d_out, B * T * C * sizeof(float));

    // 创建CUDA事件
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 记录开始时间
    hipEventRecord(start);

    //拷贝数据到显存
    hipMemcpy(d_inp, inp, B * T * C * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, C * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, C * sizeof(float), hipMemcpyHostToDevice);

    layernorm_kernel<<<grid_size, block_size>>>(d_inp, d_mean, d_rstd, d_weight, d_bias, d_out, B, T, C);
    CUDA_CHECK(hipGetLastError());

    hipMemcpy(out, d_out, B * T * C * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(rstd, d_rstd, B * T * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mean, d_mean, B * T * sizeof(float), hipMemcpyDeviceToHost);

    // 记录结束时间
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // 计算执行时间(毫秒)
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // 计算带宽 (GB/s)
    float totalBytes = 2 * B * T * C * sizeof(float) + 2 * B * T * sizeof(float) + 2 * C * sizeof(float);
    float bandwidth = totalBytes / (milliseconds * 1e6);
    // 输出结果
    printf("GPU耗时: block_size %4d | time %.4f ms | bandwidth %.2f GB/s\n",
           block_size, milliseconds, bandwidth);

    hipFree(d_inp);
    hipFree(d_mean);
    hipFree(d_rstd);
    hipFree(d_weight);
    hipFree(d_bias);
    hipFree(d_out);
}

void layernorm_cpu(const float* inp, float* mean, float* rstd, const float* weight, const float* bias, float* out
    , int B, int T, int C) {
    float eps = 1e-5f;
    for (int b = 0;b < B; b++) {
        //按照word维度拆分，计算每个word的均值和方差
        for (int t = 0; t < T; t++) {
            const float* t_input = inp + b * T * C + t * C;
            //计算均值
            float m = 0.0f;
            for (int c = 0; c < C; c++) {
                m += t_input[c];
            }
            m = m / C;
            //计算方差
            float v = 0.0f;
            for (int c = 0; c < C; c++) {
                float x_i = t_input[c] - m;
                v += x_i * x_i;
            }
            v = v / C;
            //计算标准差的倒数
            float s = 1.0f / sqrtf(v + eps);
            //计算结果输出
            float* t_output = out + b * T * C + t * C;
            for (int c = 0; c < C; c++) {
                float n = s * (t_input[c] - m);
                t_output[c] = weight[c] * n + bias[c];
            }
            //记录均值和标准差的倒数
            mean[b * T + t] = m;
            rstd[b * T + t] = s;
        }
    }
}

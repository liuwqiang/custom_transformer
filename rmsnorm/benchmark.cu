#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <numeric>
#include <algorithm>
#include "rmsnorm.cuh"

// Benchmark 测试类
class Benchmark {
public:
    // 运行单轮测试
    template <typename Fn, typename... Args>
    static double run_once(Fn&& kernel, Args&&... args) {
        return kernel(std::forward<Args>(args)...);
    }

    // 运行多轮测试并统计结果
    template <typename Fn, typename... Args>
    static void run_benchmark(int rounds, size_t totalBytes, Fn&& kernel, Args&&... args) {
        std::vector<double> timings;
        timings.reserve(rounds);

        //首次启动不计入benchmark内
        run_once(kernel, std::forward<Args>(args)...);

        // 正式测试
        for (size_t i = 0; i < rounds; ++i) {
            double t = run_once(kernel, std::forward<Args>(args)...);
            timings.push_back(t);
            std::cout << "Round " << i + 1 << ": " << t << " ms" << std::endl;
        }

        // 计算统计结果
        double sum = std::accumulate(timings.begin(), timings.end(), 0.0);
        double mean = sum / rounds;

        // 计算带宽 (GB/s)
        double bandwidth = totalBytes / (mean * 1e6);
        double sq_sum = std::inner_product(timings.begin(), timings.end(), timings.begin(), 0.0);
        double stddev = std::sqrt(sq_sum / rounds - mean * mean);

        printf("\n=== Benchmark Results ===\n");
        printf("Rounds: %d\n", rounds);
        printf("Average: %.3f ms\n", mean);
        printf("StdDev: %.3f ms\n", stddev);
        printf("Min: %.3f ms\n", *std::min_element(timings.begin(), timings.end()));
        printf("Max: %.3f ms\n", *std::max_element(timings.begin(), timings.end()));
        printf("Bandwidth: %.2f GB/s\n", bandwidth);
    }
};

void rand(float* elements, int size) {
    for(int i = 0; i < size; i++){
        elements[i] = (float)rand() / RAND_MAX;
    }
}

float rms_norm_base_gpu(float* inp, float* weight, float* out, int B, int T, int C) {
    //计算grid
    double N = B * T * C;
    int grid_size = ceil((N + block_size - 1)/ block_size);

    //分配显存
    float* d_inp;
    hipMalloc(&d_inp, B * T * C * sizeof(float));

    float* d_weight;
    hipMalloc(&d_weight, C * sizeof(float));

    float* d_out;
    hipMalloc(&d_out, B * T * C * sizeof(float));

    //拷贝数据到显存
    hipMemcpy(d_inp, inp, B * T * C * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, C * sizeof(float), hipMemcpyHostToDevice);

    // 创建CUDA事件
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // 记录开始时间
    hipEventRecord(start);

    rms_norm_kernel<<<grid_size, block_size>>>(d_inp, d_weight, d_out, B, T, C);

    // 记录结束时间
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, B * T * C * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_inp);
    hipFree(d_weight);
    hipFree(d_out);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return milliseconds;
}


int main() {
    int B = 1024, T = 768, C = 128, round = 10;
    float* inp = (float*) malloc(B * T * C * sizeof(float));
    rand(inp, B * T * C);

    float* weight = (float*) malloc(C * sizeof(float));
    rand(weight, C);

    float* out = (float*) malloc(B * T * C * sizeof(float));

    const size_t totalBytes = B * T * C * 2 * sizeof(float) + C * sizeof(float);
    Benchmark::run_benchmark(
            round, totalBytes,
            rms_norm_base_gpu,
            inp, weight, out, B, T, C
    );
    return 0;
}
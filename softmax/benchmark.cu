#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <numeric>
#include <algorithm>
#include "softmax.cuh"

// Benchmark 测试类
class Benchmark {
public:
    // 运行单轮测试
    template <typename Fn, typename... Args>
    static double run_once(Fn&& kernel, Args&&... args) {
        return kernel(std::forward<Args>(args)...);
    }

    // 运行多轮测试并统计结果
    template <typename Fn, typename... Args>
    static void run_benchmark(int rounds, size_t totalBytes, int block_size, Fn&& kernel, Args&&... args) {
        std::vector<double> timings;
        timings.reserve(rounds);

        //首次启动不计入benchmark内
        run_once(kernel, std::forward<Args>(args)...);

        // 正式测试
        for (size_t i = 0; i < rounds; ++i) {
            double t = run_once(kernel, std::forward<Args>(args)...);
            timings.push_back(t);
            std::cout << "Round " << i + 1 << ": " << t << " ms" << std::endl;
        }

        // 计算统计结果
        double sum = std::accumulate(timings.begin(), timings.end(), 0.0);
        double mean = sum / rounds;

        // 计算带宽 (GB/s)
        double bandwidth = totalBytes / (mean * 1e6);

        double sq_sum = std::inner_product(timings.begin(), timings.end(), timings.begin(), 0.0);
        double stddev = std::sqrt(sq_sum / rounds - mean * mean);

        printf("\n=== Benchmark Results ===\n");
        printf("BlockSize: %d\n", block_size);
        printf("Rounds: %d\n", rounds);
        printf("Average: %.3f ms\n", mean);
        printf("StdDev: %.3f ms\n", stddev);
        printf("Min: %.3f ms\n", *std::min_element(timings.begin(), timings.end()));
        printf("Max: %.3f ms\n", *std::max_element(timings.begin(), timings.end()));
        printf("Bandwidth: %.2f GB/s", bandwidth);
    }
};

void rand(float* elements, int size) {
    for(int i = 0; i < size; i++){
        elements[i] = (float)rand() / RAND_MAX;
    }
}

float softmax_gpu_base(const float *inp, float *out, int B, int T, int C, int block_size) {
    //计算grid
    double N = B * T;
    int grid_size = ceil((N + block_size - 1)/ block_size);

    //分配显存
    float* d_inp;
    hipMalloc(&d_inp, B * T * C * sizeof(float));
    float* d_out;
    hipMalloc(&d_out, B * T * C * sizeof(float));

    //拷贝数据到显存
    hipMemcpy(d_inp, inp, B * T * C * sizeof(float), hipMemcpyHostToDevice);

    // 创建CUDA事件
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // 记录开始时间
    hipEventRecord(start);

    softmax_kernel<<<grid_size, block_size>>>(d_inp, d_out, B, T, C);

    // 记录结束时间
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, B * T * C * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_inp);
    hipFree(d_out);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return milliseconds;
}

float softmax_gpu_v1(const float *inp, float *out, int B, int T, int C, int block_size) {
    //计算grid
    double N = B * T;

    //分配显存
    float* d_inp;
    hipMalloc(&d_inp, B * T * C * sizeof(float));
    float* d_out;
    hipMalloc(&d_out, B * T * C * sizeof(float));

    //拷贝数据到显存
    hipMemcpy(d_inp, inp, B * T * C * sizeof(float), hipMemcpyHostToDevice);

    // 创建CUDA事件
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // 记录开始时间
    hipEventRecord(start);

    softmax_kernel<<<N, block_size, (C + block_size + 1/ block_size) * sizeof(float)>>>(d_inp, d_out, B, T, C);

    // 记录结束时间
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, B * T * C * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_inp);
    hipFree(d_out);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return milliseconds;
}

float softmax_gpu_v2(const float *inp, float *out, int B, int T, int C, int block_size) {
    //计算grid
    double N = B * T;
    int grid_size = ceil((N + block_size - 1)/ block_size);

    //分配显存
    float* d_inp;
    hipMalloc(&d_inp, B * T * C * sizeof(float));
    float* d_out;
    hipMalloc(&d_out, B * T * C * sizeof(float));

    //拷贝数据到显存
    hipMemcpy(d_inp, inp, B * T * C * sizeof(float), hipMemcpyHostToDevice);

    // 创建CUDA事件
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // 记录开始时间
    hipEventRecord(start);

    softmax_kernel<<<grid_size, block_size>>>(d_inp, d_out, B, T, C);

    // 记录结束时间
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, B * T * C * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_inp);
    hipFree(d_out);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return milliseconds;
}

int main() {
    int B = 64, T = 1024, C = 768, block_size = 128, round = 10;
    float* inp = (float*) malloc(B * T * C * sizeof(float));
    rand(inp, B * T * C);

    float* out = (float*) malloc(B * T * C * sizeof(float));

    // 计算总数据量 (bytes)
    size_t totalBytes = (B * T * C * sizeof(float) * 2); // inp + out

    Benchmark::run_benchmark(
            round, totalBytes,block_size,
            softmax_gpu_v2,
            inp, out,
            B, T, C, block_size
    );

    return 0;
}